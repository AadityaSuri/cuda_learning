
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define N 250000

void __global__ add_cuda(long int *a, long int *b, long int *c) {
    long int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

void add_normal(long int *a, long int *b, long int *c) {
    for (long int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    long int a[N], b[N], c_cuda[N], c_normal[N];
    long int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(long int));
    hipMalloc((void**)&dev_b, N * sizeof(long int));
    hipMalloc((void**)&dev_c, N * sizeof(long int));

    for (long int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a, a, N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(long int), hipMemcpyHostToDevice);

    auto t1 = std::chrono::high_resolution_clock::now();
    add_cuda<<<N, 1>>>(dev_a, dev_b, dev_c);
    auto t2 = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
    std::cout << "Time taken by CUDA: " << duration << " microseconds" << std::endl;

    hipMemcpy(c_cuda, dev_c, N * sizeof(long int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < N; i++) {
    //     std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    // }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    auto t3 = std::chrono::high_resolution_clock::now();
    add_normal(a, b, c_normal);
    auto t4 = std::chrono::high_resolution_clock::now();

    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();
    std::cout << "Time taken by normal: " << duration2 << " microseconds" << std::endl;


    return 0;
}